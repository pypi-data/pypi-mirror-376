// C/C++
#include <cstdio>

// cuda
#include <hip/hip_runtime.h>

// snap
#include "cuda_utils.h"

int checkCudaError(const char *msg) {
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error occured in : %s\n", msg);
    printf("Cause: %s\n", hipGetErrorString(err));
    return 1;
  }

  return 0;
}
