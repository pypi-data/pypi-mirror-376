/* -------------------------------------------------------------------------- */
// Failed attempt to find monoliths with cuda.
//
// I'm somewhat sure the lack of memory locality between different seeds and
// the large-ish fractal perlin structs for the tiny cores killed performance.
//
// Maybe it's possible to fix it, or perhaps certain computations are just
// better suited for the CPU even if embarrassingly parallel. This was my first
// time toying with cuda, chances are I did something wrong, PRs are welcome
// to improve this code for the endeavor to find the largest monoliths :^)
//
// Compiling and running:
// - Install CUDA from your package manager (nvcc), have it on path, good luck
//
// Either manually:
//   - Run: `meson setup --buildtype release ./build --reconfigure`
//   - Run: `ninja -C ./build`
//   - Run: `./build/monolithium`
//
// Or install [uv](https://docs.astral.sh/uv/):
//   - Run: `uv run monocuda`
/* -------------------------------------------------------------------------- */

#include <cmath>
#include <cstdint>
#include <cstdio>
#include <unistd.h>

#include <hip/hip_runtime.h>

#define Gpu __device__
#define Cpu __host__

// Read Cargo.toml for more info!
#define SKIP_REJECTION 1

// Read Cargo.toml for more info!
#define SKIP_TABLE 1

/* -------------------------------------------------------------------------- */
// Utility functions

Gpu inline float fade(float t) {
    return t * t * t * (t * (t * 6.0 - 15.0) + 10.0);
}

Gpu inline float lerp(float t, float a, float b) {
    // return a + t * (b - a);
    return fma(t, b - a, a);
}

Gpu const float GRAD_LOOKUP[16][3] = {
    { 1.0f,  1.0f,  0.0f}, //  0:  x + y
    {-1.0f,  1.0f,  0.0f}, //  1: -x + y
    { 1.0f, -1.0f,  0.0f}, //  2:  x - y
    {-1.0f, -1.0f,  0.0f}, //  3: -x - y
    { 1.0f,  0.0f,  1.0f}, //  4:  x + z
    {-1.0f,  0.0f,  1.0f}, //  5: -x + z
    { 1.0f,  0.0f, -1.0f}, //  6:  x - z
    {-1.0f,  0.0f, -1.0f}, //  7: -x - z
    { 0.0f,  1.0f,  1.0f}, //  8:  y + z
    { 0.0f, -1.0f,  1.0f}, //  9: -y + z
    { 0.0f,  1.0f, -1.0f}, // 10:  y - z
    { 0.0f, -1.0f, -1.0f}, // 11: -y - z
    { 1.0f,  1.0f,  0.0f}, // 12:  y + x
    { 0.0f, -1.0f,  1.0f}, // 13: -y + z
    {-1.0f,  1.0f,  0.0f}, // 14:  y - x
    { 0.0f, -1.0f, -1.0f}, // 15: -y - z
};

Gpu inline float grad(uint8_t hash, float x, float y, float z) {
    const float* weights = GRAD_LOOKUP[hash & 0x0F];
    return fma(weights[0], x, fma(weights[1], y, weights[2] * z));
}

/* -------------------------------------------------------------------------- */
// Java RNG implementation

constexpr int64_t M = (1LL << 48) - 1;
constexpr int64_t A = 0x5DEECE66DLL;
constexpr int64_t C = 11LL;

constexpr double F64_DIV = (1ULL << 53);

struct JavaRNG {
    int64_t state;

    Gpu inline JavaRNG(uint64_t seed) {
        this->state = ((int64_t) seed ^ A) & M;
    }

    /// Roll the state, same effect as ignoring a `.next()` call
    Gpu inline void step() {
        this->state = (this->state * A + C) & M;
    }

    /// Rolls the state and returns N low bits
    Gpu inline int32_t next(uint8_t bits) {
        this->step();
        return (int32_t) (this->state >> (48 - bits));
    }

    Gpu inline int32_t next_i32_bound(int32_t max) {
        if (__popc(max) == 1) {
            return (int32_t)(((int64_t) max * (int64_t) this->next(31)) >> 31);
        }

        int32_t next = this->next(31);
        int32_t take = next % max;

        #if SKIP_REJECTION
        #else
            while (next - take + max - 1 < 0) {
                next = this->next(31);
                take = next % max;
            }
        #endif

        return take;
    }

    Gpu inline double next_f64() {
        int64_t high = ((int64_t) this->next(26)) << 27;
        int64_t low  =  (int64_t) this->next(27);
        return (double)(high | low) / F64_DIV;
    }
};

/* -------------------------------------------------------------------------- */

struct __align__(4) PerlinNoise {
    alignas(4) uint8_t map[256];
    float xoff;
    float yoff;
    float zoff;

    Gpu void init(JavaRNG* rng) {
        this->xoff = (float) (rng->next_f64() * 256.0);
        this->yoff = (float) (rng->next_f64() * 256.0);
        this->zoff = (float) (rng->next_f64() * 256.0);

        // Start a new 'arange' array
        for (int i=0; i<256; i++) {
            this->map[i] = i;
        }

        uint8_t temp;

        // Shuffle the first half
        for (int a=0; a<256; a++) {
            int b = a + rng->next_i32_bound(256 - a);
            temp = this->map[a];
            this->map[a] = this->map[b];
            this->map[b] = temp;
        }
    }

    Gpu inline float get_map(int index) {
        return this->map[index & 0xFF];
    }

    /// Sample the noise at a given coordinate
    /// - Note: For monoliths, y is often 0.0
    Gpu float sample(float x, float y, float z) {
        x += this->xoff;
        y += this->yoff;
        z += this->zoff;

        // Convert to grid coordinates (512 length)
        int xi = __float2int_rd(x) & 0xFF;
        int yi = __float2int_rd(y) & 0xFF;
        int zi = __float2int_rd(z) & 0xFF;

        // Get the fractional parts
        float xf = x - floor(x);
        float yf = y - floor(y);
        float zf = z - floor(z);

        // Smoothstep-like factors
        float u = fade(xf);
        float v = fade(yf);
        float w = fade(zf);

        // Get the hash values for the corners
        int a  = this->get_map(xi + 0 + 0);
        int aa = this->get_map(yi + a + 0);
        int ab = this->get_map(yi + a + 1);
        int b  = this->get_map(xi + 0 + 1);
        int ba = this->get_map(yi + b + 0);
        int bb = this->get_map(yi + b + 1);

        return lerp(w,
            lerp(v,
                lerp(u, grad(this->get_map(aa + zi), xf, yf, zf),
                        grad(this->get_map(ba + zi), xf - 1.0, yf, zf)),
                lerp(u, grad(this->get_map(ab + zi), xf, yf - 1.0, zf),
                        grad(this->get_map(bb + zi), xf - 1.0, yf - 1.0, zf))
            ),
            lerp(v,
                lerp(u, grad(this->get_map(aa + zi + 1), xf, yf, zf - 1.0),
                        grad(this->get_map(ba + zi + 1), xf - 1.0, yf, zf - 1.0)),
                lerp(u, grad(this->get_map(ab + zi + 1), xf, yf - 1.0, zf - 1.0),
                        grad(this->get_map(bb + zi + 1), xf - 1.0, yf - 1.0, zf - 1.0))
            )
        );
    }

    /// Roll the generator state that would have created a PerlinNoise
    /// - Fast way around without as many memory operations
    Gpu static void discard(JavaRNG* rng, int count) {

        // Gotta love magic numbers!
        #if SKIP_TABLE
            // Note: Only for (count == 48)
            rng->state *= 249870891710593LL;
            rng->state += 44331453843488LL;
            rng->state &= M;
        #else
            for (int i=0; i<count; i++) {

                // Coordinates f64 offsets
                for (int j=0; j<3; j++) {
                    rng->next_f64();
                }

                // Permutations swapping
                for (int max=256; max>=1; max--) {
                    rng->next_i32_bound(max);
                }
            }
        #endif
    }
};

/* -------------------------------------------------------------------------- */

template<int OCTAVES> struct FractalPerlin {
    PerlinNoise noise[OCTAVES];

    Gpu void init(JavaRNG* rng) {
        for (int i=0; i<OCTAVES; i++) {
            this->noise[i].init(rng);
        }
    }

    Gpu inline int octave_scale(int index) {
        return (1 << index);
    }

    Gpu float sample(float x, float y, float z) {
        float sum = 0.0f;
        for (int i=0; i<OCTAVES; i++) {
            int s = this->octave_scale(i);
            sum += this->noise[i].sample(x/s, y/s, z/s) * s;
        }
        return sum;
    }
};

/* -------------------------------------------------------------------------- */

struct World {
    FractalPerlin<10> hill;
    FractalPerlin<16> depth;

    Gpu void init(uint64_t seed) {
        JavaRNG rng(seed);

        // Skip 48 generators priorly used elsewhere
        PerlinNoise::discard(&rng, 48);

        this->hill.init(&rng);
        this->depth.init(&rng);
    }

    // Check if a given coordinate is part of a monolith
    Gpu bool is_monolith(int64_t x, int64_t z) {
        float depth = this->depth.sample(
            (float) (x/4) * 100.0, 0.0,
            (float) (z/4) * 100.0
        );

        if (fabs(depth) < 8000.0)
            return false;

        float hill = this->hill.sample(
            (float) (x/4) * 1.0, 0.0,
            (float) (z/4) * 1.0
        );

        return hill < -512.0;
    }

    Gpu bool around_spawn(int64_t radius, int64_t step) {
        for (int x=-radius; x<=radius; x+=step) {
            for (int z=-radius; z<=radius; z+=step) {
                if (this->is_monolith(x, z)) {
                    return true;
                }
            }
        }
        return false;
    }
};

/* -------------------------------------------------------------------------- */

__global__ void get_monoliths_world_per_block(
    int start, int seeds,
    float* results
) {
    int idx  = threadIdx.x;
    int dim  = blockDim.x;
    int blk  = blockIdx.x;
    int seed = start + blk;

    __shared__ World world;

    if (threadIdx.x == 0) {
        world.init(seed);

        if (blockIdx.x % 1000 == 0)
            printf("Block %d seed %d\n", blk, seed);

        if (!world.around_spawn(200, 100))
            return;
    }

    __syncthreads();

    int64_t side = 4096;
    int64_t step = 32;
    float   area = 0;

    // Each thread sums its strip
    for (int64_t x=-side+idx; x<=side; x+=step*dim) {
        for (int64_t z=-side; z<=side; z+=step) {
            area += world.is_monolith(x, z) ? step*step : 0.0f;
        }
    }

    atomicAdd(&results[blk], area);
}

__global__ void get_monoliths_world_per_thread(
    int start, int seeds,
    float* results
) {
    int tdx  = threadIdx.x;
    int dim  = blockDim.x;
    int blk  = blockIdx.x;
    int tid  = (blk * dim) + tdx;
    int seed = start + tid;

    World world;
    world.init(seed);

    if (tid % 10000 == 0)
        printf("Block %d seed %d\n", blk, seed);

    if (!world.around_spawn(200, 200))
        return;

    constexpr int64_t side = 256;
    constexpr int64_t step = 4;
    constexpr float step_area = (step * step);

    for (int64_t x=-side; x<=side; x+=step) {
        for (int64_t z=-side; z<=side; z+=step) {
            if (world.is_monolith(x, z)) {
                results[tid] += step_area;
            }
        }
    }
}

/* -------------------------------------------------------------------------- */

enum Variant {
    WORLD_PER_THREAD,
    WORLD_PER_BLOCK,
};

int main() {
    int start  = 0;
    int seeds  = 10000000;
    int thread = 64;

    float* d_results;
    hipMalloc(&d_results, seeds * sizeof(float));

    Variant variant = WORLD_PER_THREAD;
    // Variant variant = WORLD_PER_BLOCK;

    if (variant == WORLD_PER_THREAD) {
        int blocks = (seeds + thread - 1) / thread;
        get_monoliths_world_per_thread<<<blocks, thread>>>(start, seeds, d_results);
    } else if (variant == WORLD_PER_BLOCK) {
        int blocks = seeds;
        get_monoliths_world_per_block<<<blocks, thread>>>(start, seeds, d_results);
    }

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Fixme: Make a iterative loop searching chunks of N seeds
    float* results = new float[seeds];
    hipMemcpy(results, d_results, seeds * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_results);

    // Print findings
    for (int i=0; i<seeds; i++) {
        if (results[i] > 0) {
            printf("Seed %llu area: %f\n", start + i, results[i]);
        }
    }

    return 0;
}