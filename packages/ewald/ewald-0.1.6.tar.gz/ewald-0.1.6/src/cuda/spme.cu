// spme_fft.cu
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <cstdio>

static __global__ void scale_c(hipfftComplex* a, size_t n, float s){
    size_t i = blockIdx.x * size_t(blockDim.x) + threadIdx.x;
    if (i < n) { a[i].x *= s; a[i].y *= s; }
}

struct PlanWrap {
    hipfftHandle plan;
    size_t n_per_grid;
    hipStream_t stream;
};

extern "C"
void* spme_make_plan_c2c(int nx, int ny, int nz, void* cu_stream) {
    auto* w = new PlanWrap();
    w->n_per_grid = size_t(nx) * ny * nz;
    w->stream = reinterpret_cast<hipStream_t>(cu_stream);

    hipfftResult r = hipfftPlan3d(&w->plan, nx, ny, nz, HIPFFT_C2C);
    if (r != HIPFFT_SUCCESS) { printf("hipfftPlan3d err=%d\n", int(r)); delete w; return nullptr; }

    r = hipfftSetStream(w->plan, w->stream);
    if (r != HIPFFT_SUCCESS) { printf("hipfftSetStream err=%d\n", int(r)); hipfftDestroy(w->plan); delete w; return nullptr; }

    return w;
}

extern "C"
void spme_exec_inverse_3_c2c(void* plan, hipfftComplex* exk, hipfftComplex* eyk, hipfftComplex* ezk) {
    auto* w = reinterpret_cast<PlanWrap*>(plan);
    if (!w) return;
    hipfftExecC2C(w->plan, exk, exk, HIPFFT_BACKWARD);
    hipfftExecC2C(w->plan, eyk, eyk, HIPFFT_BACKWARD);
    hipfftExecC2C(w->plan, ezk, ezk, HIPFFT_BACKWARD);
}

extern "C"
void spme_scale_c2c(hipfftComplex* data, size_t n, float scale, void* cu_stream) {
    auto stream = reinterpret_cast<hipStream_t>(cu_stream);
    int threads = 256;
    int blocks  = int((n + threads - 1) / threads);
    scale_c<<<blocks, threads, 0, stream>>>(data, n, scale);
}

extern "C"
void spme_destroy_plan(void* plan) {
    auto* w = reinterpret_cast<PlanWrap*>(plan);
    if (!w) return;
    hipfftDestroy(w->plan);
    delete w;
}
